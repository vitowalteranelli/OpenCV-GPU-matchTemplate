#include "hip/hip_runtime.h"
#include "librerie.h" 
#include "MatchTemplate_AP.h"

texture<Ncv8u,  1, hipReadModeElementType> tex8u;
//texture<Ncv32u, 1, hipReadModeElementType> tex32u;
//texture<uint2,  1, hipReadModeElementType> tex64u;

const Ncv32u NUM_SCAN_THREADS = 256;
const Ncv32u LOG2_NUM_SCAN_THREADS = 8;
NCV_CT_ASSERT(K_WARP_SIZE == 32);


template<class T_in, class T_out>
struct _scanElemOp
{
    template<bool tbDoSqr>
    static inline __host__ __device__ T_out scanElemOp(T_in elem)
    {
        return scanElemOp( elem, Int2Type<(int)tbDoSqr>() );
    }

private:

    template <int v> struct Int2Type { enum { value = v }; };

    static inline __host__ __device__ T_out scanElemOp(T_in elem, Int2Type<0>)
    {
        return (T_out)elem;
    }

    static inline __host__ __device__ T_out scanElemOp(T_in elem, Int2Type<1>)
    {
        return (T_out)(elem*elem);
    }
};


static Ncv32u getPaddedDimension(Ncv32u dim, Ncv32u elemTypeSize, Ncv32u allocatorAlignment)
{
    Ncv32u alignMask = allocatorAlignment-1;
    Ncv32u inverseAlignMask = ~alignMask;
    Ncv32u dimBytes = dim * elemTypeSize;
    Ncv32u pitch = (dimBytes + alignMask) & inverseAlignMask;
    Ncv32u PaddedDim = pitch / elemTypeSize;
    return PaddedDim;
}


template<class T>
inline __device__ T readElem(T *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs);

template<>
inline __device__ Ncv8u readElem<Ncv8u>(Ncv8u *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return tex1Dfetch(tex8u, texOffs + srcStride * blockIdx.x + curElemOffs);
}

template<>
inline __device__ Ncv32u readElem<Ncv32u>(Ncv32u *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return d_src[curElemOffs];
}

template<>
inline __device__ Ncv32f readElem<Ncv32f>(Ncv32f *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return d_src[curElemOffs];
}


template <class T>
inline __device__ T warpScanInclusive(T idata, volatile T *s_Data)
{
#if __CUDA_ARCH__ >= 300
    const unsigned int laneId = cv::gpu::device::Warp::laneId();

    // scan on shuffl functions
    #pragma unroll
    for (int i = 1; i <= (K_WARP_SIZE / 2); i *= 2)
    {
        const T n = cv::gpu::device::shfl_up(idata, i);
        if (laneId >= i)
              idata += n;
    }

    return idata;
#else
    Ncv32u pos = 2 * threadIdx.x - (threadIdx.x & (K_WARP_SIZE - 1));
    s_Data[pos] = 0;
    pos += K_WARP_SIZE;
    s_Data[pos] = idata;

    s_Data[pos] += s_Data[pos - 1];
    s_Data[pos] += s_Data[pos - 2];
    s_Data[pos] += s_Data[pos - 4];
    s_Data[pos] += s_Data[pos - 8];
    s_Data[pos] += s_Data[pos - 16];

    return s_Data[pos];
#endif
}


template <class T>
inline __device__ T warpScanExclusive(T idata, volatile T *s_Data)
{
    return warpScanInclusive(idata, s_Data) - idata;
}


template <class T, Ncv32u tiNumScanThreads>
inline __device__ T blockScanInclusive(T idata, volatile T *s_Data)
{
    if (tiNumScanThreads > K_WARP_SIZE)
    {
        //Bottom-level inclusive warp scan
        T warpResult = warpScanInclusive(idata, s_Data);

        //Save top elements of each warp for exclusive warp scan
        //sync to wait for warp scans to complete (because s_Data is being overwritten)
        __syncthreads();
        
		if( (threadIdx.x & (K_WARP_SIZE - 1)) == (K_WARP_SIZE - 1) )
        {
            s_Data[threadIdx.x >> K_LOG2_WARP_SIZE] = warpResult;
        }

        //wait for warp scans to complete
        __syncthreads();

        if( threadIdx.x < (tiNumScanThreads / K_WARP_SIZE) )
        {
            //grab top warp elements
            T val = s_Data[threadIdx.x];
            //calculate exclusive scan and write back to shared memory
            s_Data[threadIdx.x] = warpScanExclusive(val, s_Data);
        }

        //return updated warp scans with exclusive scan results
        __syncthreads();
        
		return warpResult + s_Data[threadIdx.x >> K_LOG2_WARP_SIZE];
    }
    else
    {
        return warpScanInclusive(idata, s_Data);
    }
}


template <class T_in, class T_out, bool tbDoSqr>
__global__ void scanRows(T_in *d_src, Ncv32u texOffs, Ncv32u srcWidth, Ncv32u srcStride, T_out *d_II, Ncv32u IIstride)
{
    //advance pointers to the current line
    if (sizeof(T_in) != 1)
    {
        d_src += srcStride * blockIdx.x;
    }
    
	//for initial image 8bit source we use texref tex8u
    d_II += IIstride * blockIdx.x;

    Ncv32u numBuckets = (srcWidth + NUM_SCAN_THREADS - 1) >> LOG2_NUM_SCAN_THREADS;
    Ncv32u offsetX = 0;

    __shared__ T_out shmem[NUM_SCAN_THREADS * 2];
    __shared__ T_out carryElem;
    carryElem = 0;
    __syncthreads();

    while (numBuckets--)
    {
        Ncv32u curElemOffs = offsetX + threadIdx.x;
        T_out curScanElem;

        T_in curElem;
        T_out curElemMod;

        if (curElemOffs < srcWidth)
        {
            //load elements
            curElem = readElem<T_in>(d_src, texOffs, srcStride, curElemOffs);
        }
        curElemMod = _scanElemOp<T_in, T_out>::scanElemOp<tbDoSqr>(curElem);

        //inclusive scan
        curScanElem = blockScanInclusive<T_out, NUM_SCAN_THREADS>(curElemMod, shmem);

        if (curElemOffs <= srcWidth)
        {
            //make scan exclusive and write the bucket to the output buffer
            d_II[curElemOffs] = carryElem + curScanElem - curElemMod;
            offsetX += NUM_SCAN_THREADS;
        }

        //remember last element for subsequent buckets adjustment
        __syncthreads();
        if (threadIdx.x == NUM_SCAN_THREADS-1)
        {
            carryElem += curScanElem;
        }
        __syncthreads();
    }

    if (offsetX == srcWidth && !threadIdx.x)
    {
        d_II[offsetX] = carryElem;
    }
}


template <bool tbDoSqr, class T_in, class T_out>
NCVStatus scanRowsWrapperDevice(T_in *d_src, Ncv32u srcStride, T_out *d_dst, Ncv32u dstStride, NcvSize32u roi)
{
    hipChannelFormatDesc cfdTex;
    
	size_t alignmentOffset = 0;
    
	if (sizeof(T_in) == 1)
    {
        cfdTex = hipCreateChannelDesc<Ncv8u>();
        ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex8u, d_src, cfdTex, roi.height * srcStride), NPPST_TEXTURE_BIND_ERROR);
        if (alignmentOffset > 0)
        {
            ncvAssertCUDAReturn(hipUnbindTexture(tex8u), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex8u, d_src, cfdTex, alignmentOffset + roi.height * srcStride), NPPST_TEXTURE_BIND_ERROR);
        }
    }
    
	scanRows
        <T_in, T_out, tbDoSqr>
        <<<roi.height, NUM_SCAN_THREADS, 0, nppStGetActiveCUDAstream()>>>
        (d_src, (Ncv32u)alignmentOffset, roi.width, srcStride, d_dst, dstStride);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


NCVStatus ncvSquaredIntegralImage_device_AP(Ncv32f *d_src, Ncv32u srcStep, Ncv64f *d_dst, Ncv32u dstStep, NcvSize32u roi, INCVMemAllocator &gpuAllocator)
{
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn(gpuAllocator.memType() == NCVMemoryTypeDevice ||
                      gpuAllocator.memType() == NCVMemoryTypeNone, NPPST_MEM_RESIDENCE_ERROR);
    ncvAssertReturn((d_src != NULL && d_dst != NULL) || gpuAllocator.isCounting(), NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roi.width * sizeof(Ncv32f) &&
                    dstStep >= (roi.width + 1) * sizeof(Ncv64f) &&
					srcStep % sizeof(Ncv32f) == 0 &&
                    dstStep % sizeof(Ncv64f) == 0, NPPST_INVALID_STEP);

	srcStep /= sizeof(Ncv32f);
    dstStep /= sizeof(Ncv64f);

    Ncv32u WidthII = roi.width + 1;
    Ncv32u HeightII = roi.height + 1;
    Ncv32u PaddedWidthII32 = getPaddedDimension(WidthII, sizeof(Ncv32f), gpuAllocator.alignment());
    Ncv32u PaddedHeightII32 = getPaddedDimension(HeightII, sizeof(Ncv32f), gpuAllocator.alignment());
    Ncv32u PaddedWidthII64 = getPaddedDimension(WidthII, sizeof(Ncv64f), gpuAllocator.alignment());
    Ncv32u PaddedHeightII64 = getPaddedDimension(HeightII, sizeof(Ncv64f), gpuAllocator.alignment());
    Ncv32u PaddedWidthMax = PaddedWidthII32 > PaddedWidthII64 ? PaddedWidthII32 : PaddedWidthII64;
    Ncv32u PaddedHeightMax = PaddedHeightII32 > PaddedHeightII64 ? PaddedHeightII32 : PaddedHeightII64;

    NCVMatrixAlloc<Ncv32f> Tmp32_1(gpuAllocator, PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(Tmp32_1.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixAlloc<Ncv64f> Tmp64(gpuAllocator, PaddedWidthMax, PaddedHeightMax);
    ncvAssertReturn(Tmp64.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);

    NCVMatrixReuse<Ncv32f> Tmp32_2(Tmp64.getSegment(), gpuAllocator.alignment(), PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(Tmp32_2.isMemReused(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixReuse<Ncv64f> Tmp64_2(Tmp64.getSegment(), gpuAllocator.alignment(), PaddedWidthII64, PaddedHeightII64);
    ncvAssertReturn(Tmp64_2.isMemReused(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat;
    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

    NCV_SKIP_COND_BEGIN

    ncvStat = scanRowsWrapperDevice
		<true, Ncv32f, Ncv32f>
        (d_src, srcStep, Tmp32_2.ptr(), PaddedWidthII32, roi);
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_32f_C1R(Tmp32_2.ptr(), PaddedWidthII32*sizeof(Ncv32f), Tmp32_1.ptr(), PaddedHeightII32*sizeof(Ncv32f), NcvSize32u(WidthII, roi.height));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = scanRowsWrapperDevice
        <false, Ncv32f, Ncv64f>
        (Tmp32_1.ptr(), PaddedHeightII32, Tmp64_2.ptr(), PaddedHeightII64, NcvSize32u(roi.height, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_64f_C1R(Tmp64_2.ptr(), PaddedHeightII64*sizeof(Ncv64f), d_dst, dstStep*sizeof(Ncv64f), NcvSize32u(HeightII, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    NCV_SKIP_COND_END

    return NPPST_SUCCESS;
}